#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA

#include <hip/hip_runtime.h>
#include "kernel.h"
#include "utilityCore.hpp"

#include <cmath>
#include <cstdio>
#include <iostream>
#include <vector>

#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/device_vector.h>

#include <glm/glm.hpp>

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// Based on 27 cells * 2 for start/end, + padding so it's a multiple of 32
#define sharedGridIndicesSize 64

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3 *dev_pos_sorted;
glm::vec3 *dev_vel_sorted;
glm::vec3 *dev_vel_out;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 *arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params

  // 27 cell check experiment
  float gridCellWidthScale = 2.0f;

#if USE_27_CHECK
  gridCellWidthScale = 1.0f;
#endif

  gridCellWidth = gridCellWidthScale * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  hipMalloc((void**)&dev_pos_sorted, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos_sorted failed!");

  hipMalloc((void**)&dev_vel_sorted, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel_sorted failed!");

  hipMalloc((void**)&dev_vel_out, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel_out failed!");

  // Thrust setup
  dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
  dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeRule1Velocity(int N, int iSelf, const glm::vec3 *pos)
{
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
  glm::vec3 perceivedCenterOfMass = glm::vec3(0.0f, 0.0f, 0.0f);
  glm::vec3 outVelocity = glm::vec3(0.0f);

  const glm::vec3 selfBoidPosition = pos[iSelf];
  size_t numNeighbors = 0;

  for (size_t i = 0; i < N; i++)
  {
    if (i != iSelf && distance(pos[i], selfBoidPosition) < rule1Distance)
    {
      perceivedCenterOfMass += pos[i];
      numNeighbors++;
    }
  }

  if (numNeighbors > 0)
  {
    perceivedCenterOfMass /= numNeighbors;
    outVelocity = (perceivedCenterOfMass - selfBoidPosition) * rule1Scale;
  }

  return outVelocity;
}

__device__ glm::vec3 computeRule2Velocity(int N, int iSelf, const glm::vec3 *pos)
{
  // Rule 2: boids try to stay a distance d away from each other
  glm::vec3 oppVelocity = glm::vec3(0.0f);

  const glm::vec3 selfBoidPosition = pos[iSelf];

  for (size_t i = 0; i < N; i++)
  {
      // This logic is nearly repeated code, but I think it's fine to separate logic for better clarity.
      if (i != iSelf && distance(pos[i], selfBoidPosition) < rule2Distance)
      {
          oppVelocity -= (pos[i] - selfBoidPosition);
      }
  }

  return oppVelocity * rule2Scale;
}

__device__ glm::vec3 computeRule3Velocity(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel)
{
  // Rule 3: boids try to match the speed of surrounding boids
  glm::vec3 perceivedVelocity = glm::vec3(0.0f);
  glm::vec3 outVelocity = glm::vec3(0.0f);

  const glm::vec3 selfBoidPosition = pos[iSelf];
  size_t numNeighbors = 0;

  for (size_t i = 0; i < N; i++)
  {
      float dist = distance(pos[i], selfBoidPosition);
      if (i != iSelf && dist < rule3Distance)
      {
          perceivedVelocity += vel[i];
          numNeighbors++;
      }
  }

  if (numNeighbors > 0)
  {
    perceivedVelocity /= numNeighbors;
    outVelocity = perceivedVelocity * rule3Scale; 
  }

  return outVelocity;
}


__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  glm::vec3 rule1Velocity = computeRule1Velocity(N, iSelf, pos);
  glm::vec3 rule2Velocity = computeRule2Velocity(N, iSelf, pos);
  glm::vec3 rule3Velocity = computeRule3Velocity(N, iSelf, pos, vel);

  return rule1Velocity + rule2Velocity + rule3Velocity;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1?

  // obtain 1D boid ID
  int boidIndex = threadIdx.x + blockIdx.x * blockDim.x;

  if (boidIndex >= N) {
    return;
  }

  glm::vec3 velocityChange = computeVelocityChange(N, boidIndex, pos, vel1);
  glm::vec3 finalVelocity = vel1[boidIndex] + velocityChange;

  // Speed clamp
  if (length(finalVelocity) > maxSpeed)
  {
    finalVelocity = normalize(finalVelocity) * maxSpeed;
  }

  vel2[boidIndex] = finalVelocity;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
  // TODO-2.1
  // - Label each boid with the index of its grid cell.
  // - Set up a parallel array of integer indices as pointers to the actual
  //   boid data in pos and vel1/vel2

  int boidIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if (boidIndex < N)
  {
    // 3D grid approx = (pos - minPos) / cellWidth
    glm::vec3 gridIndex3D = pos[boidIndex];
    gridIndex3D -= gridMin;
    gridIndex3D *= inverseCellWidth;

    int gridIndex = gridIndex3Dto1D(gridIndex3D.x, gridIndex3D.y, gridIndex3D.z, gridResolution);

    gridIndices[boidIndex] = gridIndex;
    indices[boidIndex] = boidIndex;
  }
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
  int boid = blockIdx.x * blockDim.x + threadIdx.x;

  if (boid < N)
  {
    int gridCell = particleGridIndices[boid];

    if (boid >= 1 && gridCell != particleGridIndices[boid - 1])
    {
      gridCellStartIndices[gridCell] = boid;
    }

    if (boid + 1 < N && gridCell != particleGridIndices[boid + 1])
    {
      gridCellEndIndices[gridCell] = boid;
    }
  }
}

__device__ int positionToGridCell(int gridResolution, glm::vec3 pos, glm::vec3 gridMin, float inverseCellWidth) {
  glm::vec3 gridIndex3D = pos;
  gridIndex3D -= gridMin;
  gridIndex3D *= inverseCellWidth;

  return gridIndex3Dto1D(gridIndex3D.x, gridIndex3D.y, gridIndex3D.z, gridResolution);
}


__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

  int boidIndex = blockIdx.x * blockDim.x + threadIdx.x;

  if (boidIndex >= N) {
    return;
  }

  // Accumulated sums/data for our rules
  glm::vec3 rule1Velocity = glm::vec3(0.0f);
  glm::vec3 rule1CenterOfMass = glm::vec3(0.0f);
  int rule1Neighbors = 0;

  glm::vec3 rule2Velocity = glm::vec3(0.0f);

  glm::vec3 rule3PerceivedVelocity = glm::vec3(0.0f);
  int rule3Neighbors = 0;

  // Calculate min/max neighbor cell bounds
  glm::vec3 boidPosition = pos[boidIndex];
  float neighborhoodDistance = imax(rule1Distance, imax(rule2Distance, rule3Distance));

  glm::ivec3 minXYZ, maxXYZ;

#if USE_27_CHECK
  minXYZ = glm::ivec3(-1);
  maxXYZ = glm::ivec3(1);
#else
  minXYZ = (boidPosition - neighborhoodDistance - gridMin) / cellWidth;
  maxXYZ = (boidPosition + neighborhoodDistance - gridMin) / cellWidth;
#endif


  // Mathematically, we only access up to 8 cells
  for (int dz = minXYZ.z; dz <= maxXYZ.z; dz++) {
    for (int dy = minXYZ.y; dy <= maxXYZ.y; dy++) {
      for (int dx = minXYZ.x; dx <= maxXYZ.x; dx++) {
        // Access neighboring grid by min/max cells to check
        int accessedGridCell = gridIndex3Dto1D(dx, dy, dz, gridResolution);
        int startIndex = gridCellStartIndices[accessedGridCell];
        int endIndex = gridCellEndIndices[accessedGridCell];

        // Empty cell, skip
        if (startIndex == -1)
        {
          continue;
        }

        // Iterate through neighbor boids in cell
        for (int neighborBoid = startIndex; neighborBoid <= endIndex; neighborBoid++) {
          int neighborIndex = particleArrayIndices[neighborBoid];

          if (neighborIndex == boidIndex)
          {
            continue;
          }

          glm::vec3 neighborPosition = pos[neighborIndex];
          glm::vec3 currentVelocityChange = glm::vec3(0.0f);

          float distanceToNeighbor = distance(boidPosition, neighborPosition);

          // Ugly repeated code, but get it to work first and refactor later.
          if (distanceToNeighbor < rule1Distance)
          {
            rule1CenterOfMass += neighborPosition;
            rule1Neighbors++;
          }

          if (distanceToNeighbor < rule2Distance)
          {
            rule2Velocity -= (neighborPosition - boidPosition);
          }

          if (distanceToNeighbor < rule3Distance)
          {
            rule3PerceivedVelocity += vel1[neighborIndex];
            rule3Neighbors++;
          }
        }
      }
    }
  }

  if (rule1Neighbors > 0)
  {
    rule1CenterOfMass /= rule1Neighbors;
    rule1Velocity = (rule1CenterOfMass - boidPosition) * rule1Scale;
  }

  rule2Velocity *= rule2Scale;

  if (rule3Neighbors > 0)
  {
    rule3PerceivedVelocity /= rule3Neighbors;
    rule3PerceivedVelocity *= rule3Scale;
  }

  glm::vec3 netVelocityChange = rule1Velocity + rule2Velocity + rule3PerceivedVelocity;
  glm::vec3 finalVelocity = vel1[boidIndex] + netVelocityChange;

  // Speed clamp
  if (length(finalVelocity) > maxSpeed)
  {
      finalVelocity = normalize(finalVelocity) * maxSpeed;
  }

  vel2[boidIndex] = finalVelocity;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos_sorted, glm::vec3 *vel_sorted, glm::vec3 *vel_out) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

  int boidIndex = blockIdx.x * blockDim.x + threadIdx.x;

  if (boidIndex >= N) {
    return;
  }

  // Accumulated sums/data for our rules
  glm::vec3 rule1Velocity = glm::vec3(0.0f);
  glm::vec3 rule1CenterOfMass = glm::vec3(0.0f);
  int rule1Neighbors = 0;

  glm::vec3 rule2Velocity = glm::vec3(0.0f);

  glm::vec3 rule3PerceivedVelocity = glm::vec3(0.0f);
  int rule3Neighbors = 0;

  // Calculate min/max neighbor cell bounds
  glm::vec3 boidPosition = pos_sorted[boidIndex];
  float neighborhoodDistance = imax(rule1Distance, imax(rule2Distance, rule3Distance));

  glm::ivec3 minXYZ = (boidPosition - neighborhoodDistance - gridMin) / cellWidth;
  glm::ivec3 maxXYZ = (boidPosition + neighborhoodDistance - gridMin) / cellWidth;

  // Mathematically, we only access up to 8 cells
  for (int dz = minXYZ.z; dz <= maxXYZ.z; dz++) {
    for (int dy = minXYZ.y; dy <= maxXYZ.y; dy++) {
      for (int dx = minXYZ.x; dx <= maxXYZ.x; dx++) {
        // Access neighboring grid by min/max cells to check
        int accessedGridCell = gridIndex3Dto1D(dx, dy, dz, gridResolution);
        int startIndex = gridCellStartIndices[accessedGridCell];
        int endIndex = gridCellEndIndices[accessedGridCell];

        // Empty cell, skip
        if (startIndex == -1)
        {
          continue;
        }

        // Iterate through neighbor boids in cell
        for (int neighborBoid = startIndex; neighborBoid <= endIndex; neighborBoid++) {
          if (neighborBoid == boidIndex)
          {
            continue;
          }

          glm::vec3 neighborPosition = pos_sorted[neighborBoid];
          glm::vec3 currentVelocityChange = glm::vec3(0.0f);

          float distanceToNeighbor = distance(boidPosition, neighborPosition);

          if (distanceToNeighbor < rule1Distance)
          {
            rule1CenterOfMass += neighborPosition;
            rule1Neighbors++;
          }

          if (distanceToNeighbor < rule2Distance)
          {
            rule2Velocity -= (neighborPosition - boidPosition);
          }

          if (distanceToNeighbor < rule3Distance)
          {
            rule3PerceivedVelocity += vel_sorted[neighborBoid];
            rule3Neighbors++;
          }
        }
      }
    }
  }

  if (rule1Neighbors > 0)
  {
    rule1CenterOfMass /= rule1Neighbors;
    rule1Velocity = (rule1CenterOfMass - boidPosition) * rule1Scale;
  }

  rule2Velocity *= rule2Scale;

  if (rule3Neighbors > 0)
  {
    rule3PerceivedVelocity /= rule3Neighbors;
    rule3PerceivedVelocity *= rule3Scale;
  }

  glm::vec3 netVelocityChange = rule1Velocity + rule2Velocity + rule3PerceivedVelocity;
  glm::vec3 finalVelocity = vel_sorted[boidIndex] + netVelocityChange;

  // Speed clamp
  if (length(finalVelocity) > maxSpeed)
  {
      finalVelocity = normalize(finalVelocity) * maxSpeed;
  }

  vel_out[boidIndex] = finalVelocity;
}

__global__ void kernPopulateSortedPosVel(int N, glm::vec3 *pos_sorted, glm::vec3 *vel_sorted, 
    int *sorted_particleArrayIndices, glm::vec3 *pos, glm::vec3 *vel)
{
  int gridCellIndex = blockIdx.x * blockDim.x + threadIdx.x;

  if (gridCellIndex >= N) {
    return;
  }

  int sortedBoidIndex = sorted_particleArrayIndices[gridCellIndex];
  pos_sorted[gridCellIndex] = pos[sortedBoidIndex];
  vel_sorted[gridCellIndex] = vel[sortedBoidIndex];
}

__global__ void kernRestoreFinalVelocty(int N, glm::vec3 *vel_out_sorted, glm::vec3 *vel2, int *sorted_particleArrayIndices)
{
  int gridCellIndex = blockIdx.x * blockDim.x + threadIdx.x;

  if (gridCellIndex >= N) {
    return;
  }

  int sortedBoidIndex = sorted_particleArrayIndices[gridCellIndex];
  vel2[sortedBoidIndex] = vel_out_sorted[gridCellIndex];
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // blocks = divup(numObjects, blockSize)
  size_t blocks = (numObjects + blockSize - 1) / blockSize;

  // Update boid velocities
  kernUpdateVelocityBruteForce<<<blocks, blockSize>>>(numObjects, dev_pos, dev_vel1, dev_vel2);
  
  // kernUpdatePos() with vel2's data
  kernUpdatePos<<<blocks, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);
  
  // Ping-pong velocity buffers - we need to swap vel1's information with vel2's velocity
  hipMemcpy(dev_vel1, dev_vel2, sizeof(glm::vec3) * numObjects, hipMemcpyDeviceToDevice);
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed

  // Pair boid indices with a grid cell
  // Call kernComputeIndices to label particles with array index + grid index    
  size_t blocks = (numObjects + blockSize - 1) / blockSize;

  // particleArray/Grid data is on GPU
  kernComputeIndices<<<blocks, blockSize>>>(numObjects, gridSideCount, gridMinimum, 
      gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

  // Sort on GPU
  thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

  // Reset start/end pointers, this is needed so that we know certain cells have no boids
  size_t cellResetBlocks = (gridCellCount + blockSize - 1) / blockSize;
  kernResetIntBuffer<<<cellResetBlocks, blockSize>>>(gridCellCount, dev_gridCellStartIndices, -1);
  kernResetIntBuffer<<<cellResetBlocks, blockSize>>>(gridCellCount, dev_gridCellEndIndices, -1);

  // By this point, ideally, values in grid are sorted from 0 to gridCellCount - 1
  // Need to now store start and end pointers, this tells us the first index of a boid in gridCellIndex and then the last, "storing" boids in a gridCell
  kernIdentifyCellStartEnd<<<blocks, blockSize>>>(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

  // Start and end indices of each grid should now be successfully stored. We can now perform velocity updates, ideally
  kernUpdateVelNeighborSearchScattered<<<blocks, blockSize>>>(
      numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, 
      gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, 
      dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);

  // Update pos
  kernUpdatePos<<<blocks, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);
  
  // Ping-pong velocity buffers - we need to swap vel1's information with vel2's velocity
  hipMemcpy(dev_vel1, dev_vel2, sizeof(glm::vec3) * numObjects, hipMemcpyDeviceToDevice);
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
  size_t blocks = (numObjects + blockSize - 1) / blockSize;

  // particleArray/Grid data is on GPU
  kernComputeIndices<<<blocks, blockSize>>>(numObjects, gridSideCount, gridMinimum, 
      gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

  // Sort on GPU, this time include coherent pos/vel that we'll use to directly access
  thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

  // Populate the "sorted" arrays, though it's just the pos/vel mapped correctly to the sorted array indices
  kernPopulateSortedPosVel<<<blocks, blockSize>>>(numObjects, dev_pos_sorted, dev_vel_sorted, dev_particleArrayIndices, dev_pos, dev_vel1);

  // Reset start/end pointers, this is needed so that we know certain cells have no boids
  size_t cellResetBlocks = (gridCellCount + blockSize - 1) / blockSize;
  kernResetIntBuffer<<<cellResetBlocks, blockSize>>>(gridCellCount, dev_gridCellStartIndices, -1);
  kernResetIntBuffer<<<cellResetBlocks, blockSize>>>(gridCellCount, dev_gridCellEndIndices, -1);

  // By this point, ideally, values in grid are sorted from 0 to gridCellCount - 1
  // Need to now store start and end pointers, this tells us the first index of a boid in gridCellIndex and then the last, "storing" boids in a gridCell
  kernIdentifyCellStartEnd<<<blocks, blockSize>>>(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

  // Start and end indices of each grid should now be successfully stored. We can now perform velocity updates, ideally
  kernUpdateVelNeighborSearchCoherent<<<blocks, blockSize>>>(
      numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, 
      gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, 
      dev_pos_sorted, dev_vel_sorted, dev_vel_out);
  
  kernRestoreFinalVelocty<<<blocks, blockSize>>>(numObjects, dev_vel_out, dev_vel2, dev_particleArrayIndices);

  // Update pos
  kernUpdatePos<<<blocks, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);

  // Ping-pong velocity buffers - we need to swap vel1's information with vel2's velocity
  hipMemcpy(dev_vel1, dev_vel2, sizeof(glm::vec3) * numObjects, hipMemcpyDeviceToDevice);
}

__global__ void kernUpdateVelNeighborSearchCoherentSharedMem(int N, int *gridIndices_sorted, glm::vec3 *pos_sorted,
    int *gridCellStartIndices, int *gridCellEndIndices)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= N) {
    return;
  }
  
  // Populate, for the grid cell the thread is in, its start/end indices in shared memory
  // to avoid global mem lookups later. Idea is that by running our threads on the sorted
  // pos array, other nearby threads may need to access the same cells and therefore start/end indices.
  __shared__ int cellStartEndIndices[sharedGridIndicesSize];

  int minGridCell = gridIndices_sorted[blockIdx.x * blockDim.x];
  int currentSortedGridCell = gridIndices_sorted[index];
  int currentCellStartIndex = gridCellStartIndices[currentSortedGridCell];
  int currentCellEndIndex = gridCellEndIndices[currentSortedGridCell];

  // This guarantees we populate for the first thread index in a cell
  if (index == 0 || gridIndices_sorted[index] != gridIndices_sorted[index - 1])
  {
    int sharedStartEndIndex = 2 * (currentSortedGridCell - minGridCell);
    cellStartEndIndices[sharedStartEndIndex] = currentCellStartIndex;
    cellStartEndIndices[sharedStartEndIndex + 1] = currentCellEndIndex;
  }

  __syncthreads();

  // Now we do the rest of the fun stuff.
  
  // Calculate min/max neighbor cell bounds
  glm::vec3 boidPosition = pos_sorted[boidIndex];
  float neighborhoodDistance = imax(rule1Distance, imax(rule2Distance, rule3Distance));

  glm::ivec3 minXYZ = (boidPosition - neighborhoodDistance - gridMin) / cellWidth;
  glm::ivec3 maxXYZ = (boidPosition + neighborhoodDistance - gridMin) / cellWidth;

  // Mathematically, we only access up to 8 cells
  for (int dz = minXYZ.z; dz <= maxXYZ.z; dz++) {
    for (int dy = minXYZ.y; dy <= maxXYZ.y; dy++) {
      for (int dx = minXYZ.x; dx <= maxXYZ.x; dx++) {
        // Access neighboring grid by min/max cells to check
        int accessedGridCell = gridIndex3Dto1D(dx, dy, dz, gridResolution);
        int startIndex = gridCellStartIndices[accessedGridCell];
        int endIndex = gridCellEndIndices[accessedGridCell];

        // Empty cell, skip
        if (startIndex == -1)
        {
          continue;
        }

        // Iterate through neighbor boids in cell
        for (int neighborBoid = startIndex; neighborBoid <= endIndex; neighborBoid++) {
          if (neighborBoid == boidIndex)
          {
            continue;
          }

          glm::vec3 neighborPosition = pos_sorted[neighborBoid];
          glm::vec3 currentVelocityChange = glm::vec3(0.0f);

          float distanceToNeighbor = distance(boidPosition, neighborPosition);

          if (distanceToNeighbor < rule1Distance)
          {
            rule1CenterOfMass += neighborPosition;
            rule1Neighbors++;
          }

          if (distanceToNeighbor < rule2Distance)
          {
            rule2Velocity -= (neighborPosition - boidPosition);
          }

          if (distanceToNeighbor < rule3Distance)
          {
            rule3PerceivedVelocity += vel_sorted[neighborBoid];
            rule3Neighbors++;
          }
        }
      }
    }
  }
}

void Boids::stepSimulationSharedMemoryGrid(float dt)
{
  // From coherent solution, our pos index is auto sorted based on the min grid cell that exists. If we work under some
  // magical assumption that for thread N, thread N+1 will check N and N+2, and they're contained in the same warp,
  // we can have some time off by storing start/end indices of their cell and similarly other cells they'll access into
  // shared mem.

  // Block size = 128
  
  // compute indices
  // 
  // sort key/value grid cells and boid indices
  // 
  // reset buffers
  // 
  // populate awesome start/end indices
  // 
  // run the stupid neighboring search with shared memory
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleGridIndices);
  hipFree(dev_particleArrayIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);

  hipFree(dev_pos_sorted);
  hipFree(dev_vel_sorted);
  hipFree(dev_vel_out);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}